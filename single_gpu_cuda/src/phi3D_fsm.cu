#include "hip/hip_runtime.h"
/**
 * @file phi3D_fsm.c
 * @brief      Source file for 3D Phi Function that implements the parallel fast
 *             sweeping method for solving the Eikonal equation in CUDA The
 *             algorithm implemented for parallel fast sweeping method is from a
 *             paper in the Journal of Computational Physics titled "A parallel
 *             fast sweeping method for the Eikonal Equation" by Miles Detrixhe,
 *             Federic Gibou, and Chohong Min.
 *
 * @author     Shrestha, Anup
 * @date       09 OCT 2015
 *
 *
 *
 * @see        http://www.sciencedirect.com/science/article/pii/S002199911200722X
 * 
 * Copyright (c) 2016
 * Mechanical and Bio-medical Engineering Department
 * Boise State University
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include "phi3D.h"
#include "phi3D_fsm.h"

#define max(a, b) ((a > b) ? a : b)
#define min(a, b) ((a < b) ? a : b)

// Private method definitions
static void fast_sweep(Phi *p, int itr, hipPitchedPtr dPitchPtr);
static void _cudaMemcpy3D(hipPitchedPtr src, hipPitchedPtr dst,
                          hipExtent dExt, hipMemcpyKind kind);
static int iDivUp(int a, int b);

// CUDA functions
__global__ void fast_sweep_kernel(hipPitchedPtr dPitchPtr, SweepInfo s);
__device__ double solve_eikonal(double cur_dist, double minX, double minY,
                                double minZ, double dx, double dy, double dz);

/**
 * @brief         Calls the fast sweeping method a number of times specified by
 *                the iterations argument.
 *
 * @param[in,out] pf          Pointer to phi function.
 * @param[in]     iterations  Max iterations.
 */
void run_fsm(Phi *pf, int iterations) {

  int max_x = pf->x + 2;
  int max_y = pf->y + 2;
  int max_z = pf->z + 2;

  /*********************** CUDA ***********************/

  // time cuda code
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  hipPitchedPtr hostPtr =
      make_hipPitchedPtr(pf->distance, max_x * sizeof(double), max_x, max_y);

  hipPitchedPtr devicePtr;
  hipExtent dExt = make_hipExtent(max_x * sizeof(double), max_y, max_z);
  cudaCheckError();

  hipMalloc3D(&devicePtr, dExt);
  cudaCheckError();

  _cudaMemcpy3D(hostPtr, devicePtr, dExt, hipMemcpyHostToDevice);
  cudaCheckError();

  fast_sweep(pf, iterations, devicePtr);

  _cudaMemcpy3D(devicePtr, hostPtr, dExt, hipMemcpyDeviceToHost);
  cudaCheckError();

  hipFree(devicePtr.ptr);
  hipFree(hostPtr.ptr);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  printf("Parallel FSM time: %f s.\n", elapsedTime / 1000.0);
  /****************************************************/
}

/**
 * @brief         Calculates the distance field for a 3D grid by solving the
 *                Eikonal equation at each grid point using the parallel Fast
 *                Sweeping Method.
 *
 *                Sweeping Directions:
 *                (1) i = 1:I, j = 1:J, k = 1:K
 *                (2) i = I:1, j = 1:J, k = K:1
 *                (3) i = I:1, j = 1:J, k = 1:K
 *                (4) i = 1:I, j = 1:J, k = K:1
 *                (5) i = I:1, j = J:1, k = K:1
 *                (6) i = 1:I, j = J:1, k = 1:K
 *                (7) i = 1:I, j = J:1, k = K:1
 *                (8) i = I:1, j = J:1, k = 1:K
 *
 * @param[in.out] p          Pointer to phi function.
 * @param[in]     itr        Max iterations.
 * @param[in,out] dPitchPtr  Pointer to distance array in device memory.
 */
static void fast_sweep(Phi *p, int itr, hipPitchedPtr dPitchPtr) {

  // Information regarding sweeping and linear indexing
  int meshDim = 3;

  SweepInfo sw;
  sw.xDim = p->x;
  sw.dx = p->dx;
  sw.yDim = p->y;
  sw.dy = p->dy;
  sw.zDim = p->z;
  sw.dz = p->dz;

  int totalLevels = sw.xDim + sw.yDim + sw.zDim;

  // loop till the number of times to sweep
  int loop = 1;
  while (loop <= itr) {

    printf("Please wait. Sweeping...[%d/%d]\n", loop, itr);

    for (int swCount = 1; swCount <= 8; ++swCount) {
      int start = (swCount == 2 || swCount == 5 || swCount == 7 || swCount == 8)
                      ? totalLevels
                      : meshDim;
      int end = (start == meshDim) ? totalLevels + 1 : meshDim - 1;
      int incr = (start == meshDim) ? true : false;

      // sweep offset is used for translating the 3D coordinates
      // to perform sweeps from different directions
      sw.xSweepOff = (swCount == 4 || swCount == 8) ? sw.xDim + 1 : 0;
      sw.ySweepOff = (swCount == 2 || swCount == 6) ? sw.yDim + 1 : 0;
      sw.zSweepOff = (swCount == 3 || swCount == 7) ? sw.zDim + 1 : 0;

      for (int level = start; level != end;
           level = (incr) ? level + 1 : level - 1) {
        int xs = max(1, level - (sw.yDim + sw.zDim)),
            ys = max(1, level - (sw.xDim + sw.zDim));
        int xe = min(sw.xDim, level - (meshDim - 1)),
            ye = min(sw.yDim, level - (meshDim - 1));

        int xr = xe - xs + 1, yr = ye - ys + 1;
        int tth = xr * yr; // Total number of threads needed

        dim3 bs(16, 16, 1);
        if (tth < 256) {
          bs.x = xr;
          bs.y = yr;
        }
        dim3 gs(iDivUp(xr, bs.x), iDivUp(yr, bs.y), 1);

        sw.level = level;
        sw.xOffSet = xs;
        sw.yOffset = ys;

        fast_sweep_kernel << <gs, bs>>> (dPitchPtr, sw);
        hipDeviceSynchronize();
        cudaCheckError();
      }
    }
    printf("Sweeping finished!......[%d/%d]\n", loop, itr);
    ++loop;
  }
}


/**
 * @brief         CUDA kernel for the fast sweeping method.
 *
 * @param[in,out] dPitchPtr  Pointer to distance array in device memory.
 * @param[in]     s          Sweep information.
 */
__global__ void fast_sweep_kernel(hipPitchedPtr dPitchPtr, SweepInfo s) {
  int x = (blockIdx.x * blockDim.x + threadIdx.x) + s.xOffSet;
  int y = (blockIdx.y * blockDim.y + threadIdx.y) + s.yOffset;
  if (x <= s.xDim && y <= s.yDim) {
    int z = s.level - (x + y);
    if (z > 0 && z <= s.zDim) {
      int i = abs(z - s.zSweepOff);
      int j = abs(y - s.ySweepOff);
      int k = abs(x - s.xSweepOff);

      char *devPtr = (char *)dPitchPtr.ptr;
      size_t pitch = dPitchPtr.pitch;
      size_t slicePitch = pitch * (s.yDim + 2);

      double *c_row =
          (double *)((devPtr + i * slicePitch) + j * pitch); // center row
      double center = c_row[k];                              // center distance
      double left = c_row[k - 1];                            // left distance
      double right = c_row[k + 1];                           // right distance
      double up = ((double *)((devPtr + i * slicePitch) +
                              (j - 1) * pitch))[k]; // upper distance
      double down = ((double *)((devPtr + i * slicePitch) +
                                (j + 1) * pitch))[k]; // lower distance
      double front = ((double *)((devPtr + (i - 1) * slicePitch) +
                                 j * pitch))[k]; // front distance
      double back = ((double *)((devPtr + (i + 1) * slicePitch) +
                                j * pitch))[k]; // back distance

      double minX = min(left, right);
      double minY = min(up, down);
      double minZ = min(front, back);
      c_row[k] = solve_eikonal(center, minX, minY, minZ, s.dx, s.dy, s.dz);
    }
  }
}

/**
 * @brief      Solves Eikonal equation at linearized 3D index. Returns the
 *             minimum of calculated and old distance values.
 *
 * @param[in]  cur_dist  Current distance value.
 * @param[in]  minX      Minimum distance in the x-direction.
 * @param[in]  minY      Minimum distance in the y-direction.
 * @param[in]  minZ      Minimum distance in the z-direction.
 * @param[in]  dx        Spacing in the x-direction.
 * @param[in]  dy        Spacing in the y-direction.
 * @param[in]  dz        Spacing in the z-direction.
 *
 * @return     Minimum value of the solution at given index.
 */
__device__ double solve_eikonal(double cur_dist, double minX, double minY,
                                double minZ, double dx, double dy, double dz) {
  double dist_new = 0;
  double m[] = { minX, minY, minZ };
  double d[] = { dx, dy, dz };

  // sort the mins
  for (int i = 1; i < 3; i++) {
    for (int j = 0; j < 3 - i; j++) {
      if (m[j] > m[j + 1]) {
        double tmp_m = m[j];
        double tmp_d = d[j];
        m[j] = m[j + 1];
        d[j] = d[j + 1];
        m[j + 1] = tmp_m;
        d[j + 1] = tmp_d;
      }
    }
  }

  // simplifying the variables
  double m_0 = m[0], m_1 = m[1], m_2 = m[2];
  double d_0 = d[0], d_1 = d[1], d_2 = d[2];
  double m2_0 = m_0 * m_0, m2_1 = m_1 * m_1, m2_2 = m_2 * m_2;
  double d2_0 = d_0 * d_0, d2_1 = d_1 * d_1, d2_2 = d_2 * d_2;

  dist_new = m_0 + d_0;
  if (dist_new > m_1) {

    double s = sqrt(-m2_0 + 2 * m_0 * m_1 - m2_1 + d2_0 + d2_1);
    dist_new = (m_1 * d2_0 + m_0 * d2_1 + d_0 * d_1 * s) / (d2_0 + d2_1);

    if (dist_new > m_2) {

      double a =
          sqrt(-m2_0 * d2_1 - m2_0 * d2_2 + 2 * m_0 * m_1 * d2_2 - m2_1 * d2_0 -
               m2_1 * d2_2 + 2 * m_0 * m_2 * d2_1 - m2_2 * d2_0 - m2_2 * d2_1 +
               2 * m_1 * m_2 * d2_0 + d2_0 * d2_1 + d2_0 * d2_2 + d2_1 * d2_2);

      dist_new = (m_2 * d2_0 * d2_1 + m_1 * d2_0 * d2_2 + m_0 * d2_1 * d2_2 +
                  d_0 * d_1 * d_2 * a) /
                 (d2_0 * d2_1 + d2_0 * d2_2 + d2_1 * d2_2);
    }
  }

  return min(cur_dist, dist_new);
}

/*
 * Copies 3D memory from host to device and device to host.
 *
 * Arguments:
 *   hipPitchedPtr [in]  - pointer to distance array
 *   hipPitchedPtr [out] - pointer to distance array
 *   hipMemcpyKind [in]  - specifies the direction of copy
 * Returns:
 *
 */


/**
 * @brief      Copies 3D memory from host to device and device to host.
 *
 * @param[in]  src   Pointer to source distance array.
 * @param[out] dst   Pointer to destination disance array
 * @param[in]  dExt  Cuda extent.
 * @param[in]  kind  Specifies the direction of copy.
 */
static void _cudaMemcpy3D(hipPitchedPtr src, hipPitchedPtr dst,
                          hipExtent dExt, hipMemcpyKind kind) {
  hipMemcpy3DParms mcp = { 0 };

  mcp.kind = kind;
  mcp.extent = dExt;

  mcp.srcPtr = src;
  mcp.dstPtr = dst;

  hipMemcpy3D(&mcp);
  cudaCheckError();
}

/**
 * @brief      Calculates number of threads in each dimension of a thread block
 *
 * @param[in]  a
 * @param[in]  b
 *
 * @return     { description_of_the_return_value }
 */
static int iDivUp(int a, int b) {
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}
